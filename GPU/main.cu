#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : 4.cu
 Author      : joaovdd
 Version     :
 Copyright   : All rights reserved.
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "hip/hip_runtime.h"
#include ""

// #include <hip/hip_runtime_api.h>

#include "opcoesDeCompilacao_2.h"

#include "helper.cuh"

#include <hip/hip_complex.h>
#include <math.h>

//#include <time.h>
#include <chrono>

#include "benchmarks.h"

#include "sistInfo.h"
#include "sistema.cuh"

#include "PQcalc.cuh"
#include "Jacobiano.cuh"
#include "dim.cuh"
#include "newtonRaphson.cuh"
#include "impressao.h"
#include "arquivo.h"


int main(void)
{
	loadFile();
	sistema h_sistema, sistPon, * d_sistema = NULL;
	barra h_barra, barraPon, * d_barra = NULL;
	ramo h_ramo, ramoPon, * d_ramo = NULL;
	iterativo h_iterativo, iterPon, * d_iterativo = NULL;

	//if (lerTamanhos(global::arq_entrada, h_sistema)) {
	//	printf("Deu ruim...");
	//	return 1;
	//}

	//initSistema(h_sistema);
	//initBranch(h_sistema, h_ramo);
	//initBus(h_sistema, h_barra);
	//initIter(h_sistema, h_iterativo);

	//if (readCDF(global::arq_entrada, h_sistema, h_barra, h_ramo)) {
	//	printf("Erro! O arquivo de entrada não pode ser lido!\n\n");
	//	return 1;
	//}

	//InitCsrPhi(h_sistema, h_ramo);

	lerArquivoEAlocarMemoria(h_sistema, h_barra, h_ramo, h_iterativo);

	switch (global::metodo) {
	case metodo::denso:
		if (global::laconic_mode) { printf("[DENSO]\n\n"); }
		else { printf("\n\n[DENSO]\n\n"); }
		break;
	case metodo::hibridoA:
		if (global::laconic_mode) { printf("[HIBRIDO A]\n\n"); }
		else { printf("\n\n[HIBRIDO A]\n\n"); }
		break;
	case metodo::hibridoB:
		if (global::CPUsolverFlg) {
			if (global::laconic_mode) { printf("[HIBRIDO C]\n\n"); }
			else { printf("\n\n[HIBRIDO C]\n\n"); }
		}
		else {
			if (global::laconic_mode) { printf("[HIBRIDO B]\n\n"); }
			else { printf("\n\n[HIBRIDO B]\n\n"); }
		}
		break;
	case metodo::esparso:
		if (global::laconic_mode) { printf("[ESPARSO]\n\n"); }
		else { printf("\n\n[ESPARSO]\n\n"); }
		break;
	default:
		std::cout << "[METODO] METODO INVALIDO LIDO DO ARQUIVO!\n" << std::endl;
	}

	constexpr int nStreams = 4;
	hipStream_t streams[nStreams] = { nullptr };

	hipDeviceProp_t deviceProp;
	deviceProp = initGPU(); // deve ser contabilisado?

	//clock_t inicio = clock();
	//auto inicio = std::chrono::high_resolution_clock::now();
	{ BENCHMARK_GERAL
		{ BENCHMARK_ADMITANCIA
			// calcula Ybus
			switch (global::metodoDeCalculoDeYbus) {
			case metodoDeCalculoDeYbus::dnCPU:
				calcYbus(h_sistema, h_barra, h_ramo);
				break;
			case metodoDeCalculoDeYbus::spCPU:
				calcYbusSp_eficinte(h_sistema, h_barra, h_ramo);
				break;
			default:
				printf("\n\n[ERRO] calcYbus: metodo inválido!\n\n");
				return -1;
				break;
			}
		}

		if (global::verbose_mode) {
			printAll(h_sistema, h_barra, h_ramo);
		}

		// hipDeviceProp_t deviceProp;
		
		{ BENCHMARK_INITGPU
			//deviceProp = initGPU(); // deve ser contabilisado?
			//aloca espaço para as estruturas na memória global do device
			checkCudaErrors(hipMalloc(&d_sistema, sizeof(sistema)));
			checkCudaErrors(hipMalloc(&d_barra, sizeof(barra)));
			checkCudaErrors(hipMalloc(&d_ramo, sizeof(ramo)));
			checkCudaErrors(hipMalloc(&d_iterativo, sizeof(iterativo)));

			// ADICIONAL para paralelismo P Q e transferencias de memória
			if (global::streams) {
				if (global::metodo == metodo::esparso || global::metodo == metodo::hibridoB) {
					for (size_t i = 0; i < nStreams; i++)
					{
						checkCudaErrors(hipStreamCreate(&streams[i]));
					}
				}
				else if (global::metodo == metodo::denso) {
					// construa-me
				}
			}
	
			checkCudaErrors(hipGetLastError());

			//inicia variáveis alocadas dinamicamente

			sistPon = d_initSistema(h_sistema, d_sistema);
			ramoPon = d_initRamo(h_sistema, h_ramo, d_ramo);
			barraPon = d_initBarra(h_sistema, d_barra);
			iterPon = d_initIter(h_sistema, d_iterativo);
			
			checkCudaErrors(hipGetLastError());

			checkCudaErrors(hipDeviceSynchronize());
		}

		{ BENCHMARK_CUDAMEMCPY
			// transfere dados lidos para a GPU (device)

			sistemacpyH2D(h_sistema, d_sistema, sistPon);
			ramocpyH2D(h_sistema, h_ramo, d_ramo, ramoPon);
			barracpyH2D(h_sistema, h_barra, d_barra, barraPon);
			itercpyH2D(h_sistema, h_iterativo, d_iterativo, iterPon);
		}


		if (global::verbose_mode) {
			printf("V =\n");
			d_showVecf(barraPon.V, h_sistema.nB);
		}

		// Cálculo das matrizes esparsas no caso denso (para funções otimizadas)
		switch (global::metodo) {
		case metodo::hibridoA:
			d_criarYesparso(sistPon);
			// apenas para algumas abordagens híbridas 
			break;
		default:
			break;
		}

		// nR(h_sistema, h_barra, h_ramo, h_iterativo, d_sistema, d_barra, d_ramo, d_iterativo, sistPon, barraPon, ramoPon, iterPon, deviceProp);
		mNR(h_sistema, h_barra, h_ramo, h_iterativo, d_sistema, d_barra, d_ramo, d_iterativo, sistPon, barraPon, ramoPon, iterPon, deviceProp, streams);

		{ BENCHMARK_CUDAMEMCPY
			hipMemcpy(h_barra.V, barraPon.V, sizeof(float_type) * (sistPon.nB), hipMemcpyDeviceToHost);

			hipMemcpy(h_barra.theta, barraPon.theta, sizeof(float_type) * (sistPon.nB), hipMemcpyDeviceToHost);
			BENCHMARK_SYNC
		}

		//calcFluxf(h_sistema, h_barra, h_ramo, sistPon, barraPon, ramoPon, deviceProp);

		//calcFluxf_Eficiente_Sp(h_sistema, h_barra, h_ramo, h_iterativo, sistPon, barraPon, ramoPon, iterPon, deviceProp);

		{ BENCHMARK_FLUXO
			switch (global::metodo) {
			case metodo::denso:
				//calcFluxf(h_sistema, h_barra, h_ramo, sistPon, barraPon, ramoPon, deviceProp); // inieficiente
				calcFluxf_ef(h_sistema, h_barra, h_ramo, sistPon, barraPon, ramoPon, deviceProp); // inieficiente
				break;
			case metodo::hibridoA:
			case metodo::hibridoB:
			case metodo::esparso:
				calcFluxf_Eficiente_Sp(h_sistema, h_barra, h_ramo, h_iterativo, sistPon, barraPon, ramoPon, iterPon, deviceProp, streams);
				break;
			default:
				std::cout << "[calcFluxf] METODO INVALIDO LIDO DO ARQUIVO!\n" << std::endl;
			}
			BENCHMARK_SYNC
		}

		{ BENCHMARK_CUDAMEMCPY
			checkCudaErrors(hipMemcpy(h_ramo.Ppd, ramoPon.Ppd, sistPon.nL * sizeof(float_type), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(h_ramo.Pdp, ramoPon.Pdp, sistPon.nL * sizeof(float_type), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(h_ramo.Qpd, ramoPon.Qpd, sistPon.nL * sizeof(float_type), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(h_ramo.Qdp, ramoPon.Qdp, sistPon.nL * sizeof(float_type), hipMemcpyDeviceToHost));

			checkCudaErrors(hipMemcpy(h_iterativo.Pcalc, iterPon.Pcalc, sistPon.nB * sizeof(float_type), hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(h_iterativo.Qcalc, iterPon.Qcalc, sistPon.nB * sizeof(float_type), hipMemcpyDeviceToHost));

			checkCudaErrors(hipDeviceSynchronize()); // deve-se terminar o a cópia de dados antes de finalizar a contagem de tempo
		}

		//auto fim = std::chrono::high_resolution_clock::now(); //clock_t fim = clock();
	}
	//std::chrono::duration<float_type, std::milli> duracao = fim - inicio;

	//if (global::laconic_mode) {
	//	printf("%d iteracoes.\n%f ms.\n", iterPon.iteracao, duracao.count());
	//	benchmarkModePrint(iterPon, duracao);
	//}
	//else {
	//	printf("\n\nForam efetuadas %d iteracoes.\nO metodo de NR e calculo do fluxo foram feitos em %f ms.\n", iterPon.iteracao, duracao.count());
	//	impressao(h_sistema, h_barra, h_ramo, h_iterativo);
	//}

	if (!global::laconic_mode &&
		global::output_ans) {
		impressao2(h_sistema, h_barra, h_ramo, h_iterativo);
	}
	if (global::laconic_mode ||
		global::output_benchmark == output_benchmarkType::all ||
		global::output_benchmark == output_benchmarkType::file) {
		benchmarksPrintFile(iterPon);
	}
	if (!global::laconic_mode &&
		(global::output_benchmark == output_benchmarkType::screen ||
			global::output_benchmark == output_benchmarkType::all)) {
		benchmarksPrint(iterPon);
	}

	for (unsigned int i = 0; i < nStreams; i++) {
		if (streams[i]) { checkCudaErrors(hipStreamDestroy(streams[i])); }
	}

	d_finSistema (d_sistema, sistPon);
	d_finBarra (d_barra, barraPon);
	d_finRamo (d_ramo, ramoPon);
	d_finIter (d_iterativo, iterPon);

	finSistema(h_sistema);
	finBranch(h_ramo);
	finBus(h_barra);
	finIter(h_iterativo);
}
